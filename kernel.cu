#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

__global__ void vecMax_kernel(double* a, double* b, double* c, unsigned int M) {

    int i = blockDim.x*blockIdx.x + threadIdx.x;
           if(i < M) {
               double aval = a[i];
               double bval = b[i];
               c[i] = (aval > bval)?aval:bval;
           }


}

void vecMax_gpu(double* a, double* b, double* c, unsigned int M) {

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);
 double *a_d, *b_d, *c_d; 
 hipMalloc((void**) &a_d, M*sizeof(double));
 hipMalloc((void**) &b_d, M*sizeof(double)); 
 hipMalloc((void**) &c_d, M*sizeof(double));

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);
hipMemcpy(a_d, a, M*sizeof(double), hipMemcpyHostToDevice); 
hipMemcpy(b_d, b, M*sizeof(double), hipMemcpyHostToDevice);



    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);
    const unsigned int numThreadsPerBlock = 512;
    const unsigned int numBlocks = (N + numThreadsPerBlock – 1)/numThreadsPerBlock;
    vecMax_kernel <<< numBlocks, numThreadsPerBlock >>> (a_d, b_d, c_d, M);
    



    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);

    hipMemcpy(c, c_d, M*sizeof(float), hipMemcpyDeviceToHost);


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);
    hipFree(a_d); 
    hipFree(b_d); 
    hipFree(c_d);



    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}

